#include "Common.h"

void Raytracer::check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
	if (result != hipSuccess) {
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
			file << ":" << line << " '" << func << "' \n";
		// Free all device memory
		hipDeviceReset();
		exit(99);
	}
}

void Raytracer::check_curand(hipError_t result, char const* const func, const char* const file, int const line) {
	if (result != HIPRAND_STATUS_SUCCESS) {
		std::cerr << "cuRAND error = " << static_cast<unsigned int>(result) << " at " <<
			file << ":" << line << " '" << func << "' \n";
		// Free all device memory
		hipDeviceReset();
		exit(99);
	}
}