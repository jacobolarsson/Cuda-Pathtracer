#include "Common.h"

void Raytracer::check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
	if (result != hipSuccess) {
		std::cerr << "CUDA error:" << result << ": " << hipGetErrorString(result) << " at " <<
			file << ":" << line << " '" << func << "' \n";
		// Free all device memory
		hipDeviceReset();
		exit(99);
	}
}

void Raytracer::check_curand(hipError_t result, char const* const func, const char* const file, int const line) {
	if (result != HIPRAND_STATUS_SUCCESS) {
		std::cerr << "cuRAND error" << result << ": " << hipGetErrorString(result) << " at " <<
			file << ":" << line << " '" << func << "' \n";
		// Free all device memory
		hipDeviceReset();
		exit(99);
	}
}